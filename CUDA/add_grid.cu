#include "hip/hip_runtime.h"
// References: https://developer.nvidia.com/blog/even-easier-introduction-cuda/


#include <iostream>
#include <math.h>


__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+=stride) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    int N = 1 << 20;

    float *x = new float[N];
    float *y = new float[N];

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemPrefetchAsync(&x, N * sizeof(float), 0, 0);
    hipMemPrefetchAsync(&y, N * sizeof(float), 0, 0);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;

    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);
}